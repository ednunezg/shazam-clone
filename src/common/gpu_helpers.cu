#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "/usr/local/cuda/include/hipfft/hipfft.h"
#include "globals.h"
#include "gpu_helpers.h"

__device__ unsigned long computeHash_CUDA(int f1, int f2, int f3, int f4){
    
      const unsigned long FUZ_FACTOR = 3;

      if((f1 == -1 || f2==-1) || (f3==-1 || f4==-1)) return 0; //We return zero as the value of a hash in case of a silence
      unsigned long result = 0;
      result += (unsigned long) ((f4-(f4%FUZ_FACTOR)) * 100000000);
      result += (unsigned long) ((f3-(f3%FUZ_FACTOR)) * 100000);
      result += (unsigned long) ((f2-(f2%FUZ_FACTOR)) * 100);
      result += (unsigned long) (f1-(f1%FUZ_FACTOR));
      return result;
}

__global__ void tranformToHashes_CUDA(Complex * input, int numChunks, unsigned long * outputHashes){
    
    int FREQ_RANGES[5] = { 40, 80, 120, 180, 300 };

    int i = threadIdx.x + blockIdx.x*blockDim.x; //My chunk

    int firstBin = (CHUNK_SECONDS * FREQ_LOWERLIMIT)-2;
    int lastBin = (CHUNK_SECONDS  * FREQ_UPPERLIMIT)+2;
    int f1, f2, f3, f4;
    float f1val, f2val, f3val, f4val;

    f1 = -1; f2 = -1; f3 = -1 ; f4 = -1;
    f1val = 0; f2val = 0; f3val = 0 ; f4val = 0;



    if( i < numChunks)
    {
        for(int k=firstBin; k<lastBin; k++)
        {

          Complex bin = input[k+(CHUNK_SAMPLES*i)];

          int binFreq =  ((float) SAMPLE_RATE / (float) CHUNK_SAMPLES) * k;
          float binMag = sqrt( bin.re * bin.re  + bin.im * bin.im); 

          if((binFreq >= FREQ_RANGES[0] && binFreq < FREQ_RANGES[1]) && binMag > f1val ) { f1 = binFreq; f1val = binMag; }
          if((binFreq >= FREQ_RANGES[1] && binFreq < FREQ_RANGES[2]) && binMag > f2val ) { f2 = binFreq; f2val = binMag; }
          if((binFreq >= FREQ_RANGES[2] && binFreq < FREQ_RANGES[3]) && binMag > f3val ) { f3 = binFreq; f3val = binMag; }
          if((binFreq >= FREQ_RANGES[3] && binFreq < FREQ_RANGES[4]) && binMag > f4val ) { f4 = binFreq; f4val = binMag; }
        }

        unsigned long hash = computeHash_CUDA(f1, f2, f3, f4);

        //STEP 3: Hash all of the top 4 frequencies
        (outputHashes)[i] = computeHash_CUDA(f1, f2, f3, f4);

    }
}

void audioToHashes_CUDA(Complex * input, int numChunks, unsigned long ** outputHashes){

    *outputHashes = (unsigned long *) malloc( sizeof(unsigned long) * numChunks);

    /* Step 1. Perform a batch of FFTs on the complex inputs */

        //Use cuFFT, according to StackOverflow forums, it beats the benchmarks of any other FFT library available for CUDA

        Complex * d_input;
        hipMalloc((void**)&d_input, sizeof(Complex)*numChunks*CHUNK_SAMPLES);
        hipMemcpy(d_input,   input, sizeof(Complex)*numChunks*CHUNK_SAMPLES, hipMemcpyHostToDevice);

        for(int i=0; i<numChunks; ++i)
        {
            Complex * chunk = d_input + (i*CHUNK_BYTES/8);
            hipfftHandle plan;
            hipfftPlan1d(&plan, CHUNK_SAMPLES, HIPFFT_C2C, 1);
            hipfftExecC2C(plan, (hipfftComplex *)chunk, (hipfftComplex *)chunk, HIPFFT_FORWARD);
            hipfftDestroy(plan);
        }

    /* Step 2: Launch another CUDA kernel that computes a hash for each chunk of the transformed input */

      //Use transformToHashes here
      unsigned long * d_hashes;
      hipMalloc((void**)&d_hashes, sizeof(unsigned long)*numChunks);
    
      int numBlocks = numChunks/BLOCK_SIZE;
      if(numChunks%BLOCK_SIZE) numBlocks++;

      dim3 dim_grid(numBlocks,1,1);
      dim3 dim_block(BLOCK_SIZE,1,1);

      tranformToHashes_CUDA<<<dim_grid, dim_block>>>(d_input, numChunks, d_hashes);
      hipDeviceSynchronize();

      hipMemcpy( *outputHashes , d_hashes, sizeof(unsigned long)*numChunks,hipMemcpyDeviceToHost);

      return;
}