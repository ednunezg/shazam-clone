#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "/usr/local/cuda/include/hipfft/hipfft.h"
#include "globals.h"
#include "gpu_helpers.h"

__device__ unsigned long computeHash_CUDA(int f1, int f2, int f3, int f4){
    return 0x0;
}

__global__ void tranformToHashes_CUDA(Complex * input, int numChunks){
    return;
}

void audioToHashes_CUDA(Complex * input, int numChunks, unsigned long ** outputHashes){

    /* Step 1. Perform a batch of FFTs on the complex inputs */

        //Use cuFFT, according to StackOverflow forums, it beats the benchmarks of any other FFT library available for CUDA

    /* Step 2: Launch another CUDA kernel that computes a hash for each chunk of the transformed input */

        //Use transformToHashes here

    return;
}